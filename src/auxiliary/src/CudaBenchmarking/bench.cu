#ifndef _POSIX_C_SOURCE
#define _POSIX_C_SOURCE 200112L
#endif

extern "C" {

#include <stdarg.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define SECONDS 0
#define MILLIS 1
#define MICROS 2
#define NANOS 3

static const char * unitName[] = {"s", "ms", "us", "ns"};

struct cudabench {                                                   
  int                num;
  char *             name;
  int                timeUnit;
  hipEvent_t        start;
  hipEvent_t        stop;
};

void benchStart( struct cudabench *interval)
{                                                            
  hipEventCreate(&(interval->start));
  hipEventRecord(interval->start);
}                                                                        
                                                                         
void benchEnd( struct cudabench* interval)
{                                                            
  hipEventCreate(&(interval->stop));
  hipEventRecord(interval->stop);
}                         
                                               
void benchThis( void)
{                                                                        
  /* noop*/
}                                                                        

double benchRes( struct cudabench* interval)
{
  hipEventSynchronize(interval->stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, interval->start, interval->stop);
  return (double)milliseconds / 1e3;
}

char* benchName( struct cudabench* interval)
{
  size_t len = strlen(interval->name) + 1;
  char* newName = (char *)malloc(len);
  memcpy(newName, interval->name, len);
  return( newName);
}

int benchNum( struct cudabench* interval)
{
  return( interval->num);
}

char* benchUnitName( struct cudabench* interval)
{
  size_t len = strlen( unitName[interval->timeUnit]) + 1;
  char* unit_name = (char *)malloc(len);
  memcpy( unit_name, unitName[interval->timeUnit], len);
  return( unit_name);
}

int benchUnit( struct cudabench* interval)
{
  return( interval->timeUnit);
}

void benchCreate( struct cudabench** interval)                                                
{                                                                        
  /* benchGetInterval actually creates the data structure */
}     

struct cudabench* benchGetInterval_si(char * name, int num)
{                                                                        
  struct cudabench *interval;
  char* newName;
  interval = (struct cudabench*)malloc( sizeof( struct cudabench));
  interval->num = num;                                    
  newName = (char *)malloc( strlen( name) + 1);
  memcpy( newName, name, strlen( name) + 1);
  interval->name = newName;
  interval->timeUnit = SECONDS;
  return( interval);
}                                                                        
 
struct cudabench* benchGetInterval_i( int num)               
{                                                                        
  struct cudabench *interval;
  interval = benchGetInterval_si("\0", num);
  return( interval);
}                                                                        

struct cudabench* benchGetInterval_s( char *name) 
{                                                                        
  struct cudabench *interval;
  interval = benchGetInterval_si(name, -1);
  return( interval);
}                                                                        

struct cudabench* benchGetInterval_siu(char * name, int num, int timeunit)
{                                                                        
  struct cudabench *interval;
  char* newName;
  interval = (struct cudabench*)malloc( sizeof( struct cudabench));
  interval->num = num;                                    
  newName = (char *)malloc( strlen( name) + 1);
  memcpy( newName, name, strlen( name) + 1);
  interval->name = newName;
  interval->timeUnit = timeunit;
  return( interval);
}                                                                        

struct cudabench* benchGetInterval_iu( int num, int timeunit)               
{                                                                        
  struct cudabench *interval;
  interval = benchGetInterval_siu("\0", num, timeunit);
  return( interval);
}                                                                        

struct cudabench* benchGetInterval_su( char *name, int timeunit) 
{                                                                        
  struct cudabench *interval;
  interval = benchGetInterval_siu(name, -1, timeunit);
  return( interval);
}                                                                        
                                                                         
void benchDestroyInterval( struct cudabench *interval)                        
{                                                                        
  hipEventDestroy(interval->start);
  hipEventDestroy(interval->stop);
  free(interval->name);
  free(interval);                                                        
}                            
                                            
}
